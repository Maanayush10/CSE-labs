#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a, int *b, int *c)

{
    *c = *b + *a;
}



int main(void) {
    int a,b,c;
    int *d_a, *d_b, *d_c;
    // host copies of variables a, b , c;
    // device copies of variables a, b , c;
    int size = sizeof(int);

    // Allocate space for device copies of a, b,
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Setup input values

    a=3;
    b=5;

// Copy inputs to device

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a,d_b,d_c);
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    printf("Result : %d\n",c);    

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
