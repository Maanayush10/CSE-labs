/*Write a program in CUDA to add two vectors of length N using N threads*/



#include "hip/hip_runtime.h"

#include <stdio.h>

// #define N 6

__global__ void add(int *a, int *b, int *c)

{
    c[threadIdx.x] = b[threadIdx.x] + a[threadIdx.x];
}



int main(void) {
    int N;
    printf("Enter the value of N :\t");
    scanf("%d", &N);


    int a[N],b[N],c[N];
    int *d_a, *d_b, *d_c;
    // host copies of variables a, b , c;
    // device copies of variables a, b , c;
    int size = N*sizeof(int);

    // Allocate space for device copies of a, b,
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Setup input values
    printf("Enter A values :\t");
    for(int i=0; i<N; i++)
    {
        scanf("%d", &a[i]);
    }
    printf("Enter B values :\t");
    for(int i=0; i<N; i++)
    {
        scanf("%d", &b[i]);
    }
// Copy inputs to device

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<1,N>>>(d_a,d_b,d_c);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost); 
    
    //result output
    printf("\nA[i]+ b[i]= c[i]\n");
    for(int i=0; i<N; i++)
    {
        printf("\n%d + %d = %d\n", a[i], b[i], c[i]);
    }
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}